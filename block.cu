#include "hip/hip_runtime.h"
//
// Created by admin on 2021/12/4.
//
#include <iostream>
#include <hip/hip_runtime.h>
#include <string.h>
#include "kernels.cu"
#include "tensor.cu"
#include "conv_winograd_4x4_3x3.cu"
#include "conv_winograd_gpu.cu"

using namespace std;

class conv2d{
private:
    int in_channels;
    int out_channels;
    int kernel_size;
    int dialations;
    int padding;
    int strides;
    float* Weight;
    float* Bias;

public:
    conv2d(int in_c, int out_c, float* weight, float* bias, const int kernel_sz, const int dialations, const int padding, const int strides):
            in_channels(in_c),out_channels(out_c),Weight(weight),Bias(bias),kernel_size(kernel_sz),dialations(dialations),padding(padding),strides(strides){}
    //input->tensor_A; output->tensor_B
    void forward(const tensor<float>* tensor_A, tensor<float>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width;
        const int batch = tensor_A->batch;
        float *A=tensor_A->data;
        // =================================================计算输出大小
        int height_B = (height_A+2*padding-dialations*(kernel_size-1)-1)/strides + 1;
        int width_B = (width_A+2*padding-dialations*(kernel_size-1)-1)/strides + 1;

        float* B = (float*)malloc(sizeof(float)*height_B*width_B*out_channels*batch);
        tensor_B=new tensor<float>(B,width_B,height_B,out_channels,batch);

        float* d_A;
        float* d_B;
        float* d_K;
        float* d_bias;

        hipMalloc((void**)&d_A, batch * width_A * height_A * in_channels * sizeof(float));
        hipMalloc((void**)&d_B, batch * width_B * height_B * out_channels * sizeof(float));
        hipMalloc((void**)&d_K, kernel_size*kernel_size * in_channels * out_channels * sizeof(float));
        hipMalloc((void**)&d_bias, 1*1*out_channels* sizeof(float));

        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * in_channels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_K, (void*)Weight, kernel_size*kernel_size * in_channels * out_channels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_bias, (void*)Bias, 1*1 * out_channels * sizeof(float), hipMemcpyHostToDevice);

        // =================================================执行
        int nthreads = batch * width_B * height_B * out_channels;

        int num=nthreads/400+1;
        dim3 blockNum(num, 1);
        dim3 threadsPerBlock(20, 20);

        ConvolutionForward<<<blockNum, threadsPerBlock>>>(d_A, d_B, d_K, d_bias, nthreads,batch, height_A, width_A, in_channels ,height_B, width_B, out_channels,
                                                          kernel_size,kernel_size,strides,strides,padding,padding);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_B * height_B * out_channels * sizeof(float), hipMemcpyDeviceToHost);

    }
};


class conv_wino_4x4_3x3 {
private:
    int in_channels;
    int out_channels;
    int kernel_size;
    int dialations;
    int padding;
    int strides;
    float *Weight;
    float *Bias;

public:
    conv_wino_4x4_3x3(int in_c, int out_c, float *weight, float *bias, const int kernel_sz, const int dialations,
                      const int padding, const int strides) :
            in_channels(in_c), out_channels(out_c), Weight(weight), Bias(bias), kernel_size(kernel_sz),
            dialations(dialations), padding(padding), strides(strides) {}

    //input->tensor_A; output->tensor_B
    void forward(const tensor<float> *tensor_A, tensor<float> *&tensor_B) {
        const int height_A = tensor_A->height, width_A = tensor_A->width;
        const int batch = tensor_A->batch;
        float *A = tensor_A->data;
        int P = batch * ceil(height_A/4) * ceil(width_A/4);
        int tile_num = ceil(height_A/4) * ceil(width_A/4) ;
        // =================================================计算输出大小
        int height_B = (height_A + 2 * padding - dialations * (kernel_size - 1) - 1) / strides + 1;
        int width_B = (width_A + 2 * padding - dialations * (kernel_size - 1) - 1) / strides + 1;

        float *B = (float *) malloc(sizeof(float) * height_B * width_B * out_channels * batch);
        tensor_B = new tensor<float>(B, width_B, height_B, out_channels, batch);

        float *d_A;
        float *d_B;
//        float *d_K;
        float *d_bias;

        float *U = (float *) malloc(sizeof(float) * out_channels * in_channels * 36); // out_channel(4)*in_channel(2)*36
        winograd4::calc_U(Weight, U, in_channels, out_channels); // CPU function, as it can be calculated beforehand

        float *d_V, *d_U, *d_UV;

        hipMalloc((void **) &d_A, batch * width_A * height_A * in_channels * sizeof(float));
        hipMalloc((void **) &d_B, batch * width_B * height_B * out_channels * sizeof(float));
        //        hipMalloc((void**)&d_K, kernel_size*kernel_size * in_channels * out_channels * sizeof(float));
        hipMalloc((void **) &d_bias, 1 * 1 * out_channels * sizeof(float));

        hipMemcpy((void *) d_A, (void *) A, batch * width_A * height_A * in_channels * sizeof(float),
                   hipMemcpyHostToDevice);
        //        hipMemcpy((void*)d_K, (void*)Weight, kernel_size*kernel_size * in_channels * out_channels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void *) d_bias, (void *) Bias, 1 * 1 * out_channels * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc((void **) &d_V, sizeof(float) * in_channels * P * 36);
        hipMalloc((void **) &d_U, sizeof(float) * out_channels * in_channels * 36);
        hipMalloc((void **) &d_UV, sizeof(float) * out_channels * P * 36);
        hipMemcpy(d_U, U, sizeof(float) * out_channels * in_channels * 36, hipMemcpyHostToDevice);

        // =================================================执行
        winograd4::calc_V<<<dim3(batch, tile_num, in_channels), dim3(6, 6)>>>(d_A, d_V, P, batch, in_channels, height_A, width_A);
        winograd4::calc_UV<<<dim3(out_channels / 2, P / 2, 36), dim3(2, 2)>>>(d_U, d_V, d_UV, out_channels, in_channels, P);
        winograd4::calc_AtmA_bias<<<dim3(out_channels, batch, tile_num), dim3(6, 6)>>>(d_UV, d_B, d_bias, out_channels, P,
                height_B, width_B, tile_num);

        hipMemcpy((void *) tensor_B->data, (void *) d_B, batch * width_B * height_B * out_channels * sizeof(float),
                   hipMemcpyDeviceToHost);
    }
};


class conv_wino_2x2_3x3{
private:
    int in_channels;
    int out_channels;
    int kernel_size;
    int dialations;
    int padding;
    int strides;
    float* Weight;
    float* Bias;

public:
    conv_wino_2x2_3x3(int in_c, int out_c, float* weight, float* bias, const int kernel_sz, const int dialations, const int padding, const int strides):
            in_channels(in_c),out_channels(out_c),Weight(weight),Bias(bias),kernel_size(kernel_sz),dialations(dialations),padding(padding),strides(strides){}
    //input->tensor_A; output->tensor_B
    void forward(const tensor<float>* tensor_A, tensor<float>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width;
        const int batch = tensor_A->batch;
        float *A=tensor_A->data;
        int P = batch * ceil(height_A/2) * ceil(width_A/2);
        int tile_num = ceil(height_A/2) * ceil(width_A/2) ;
        // =================================================计算输出大小
        int height_B = (height_A+2*padding-dialations*(kernel_size-1)-1)/strides + 1;
        int width_B = (width_A+2*padding-dialations*(kernel_size-1)-1)/strides + 1;

        float* B = (float*)malloc(sizeof(float)*height_B*width_B*out_channels*batch);
        tensor_B=new tensor<float>(B,width_B,height_B,out_channels,batch);

        float* d_A;
        float* d_B;
//        float* d_K;
        float* d_bias;

        float* U = (float*) malloc(sizeof(float)*out_channels*in_channels*16); // out_channel(4)*in_channel(2)*36
        winograd2::calc_U(Weight, U, in_channels, out_channels); // CPU function, as it can be calculated beforehand

        hipMalloc((void**)&d_A, batch * width_A * height_A * in_channels * sizeof(float));
        hipMalloc((void**)&d_B, batch * width_B * height_B * out_channels * sizeof(float));
        //        hipMalloc((void**)&d_K, kernel_size*kernel_size * in_channels * out_channels * sizeof(float));
        hipMalloc((void**)&d_bias, 1*1*out_channels* sizeof(float));

        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * in_channels * sizeof(float), hipMemcpyHostToDevice);
        //        hipMemcpy((void*)d_K, (void*)Weight, kernel_size*kernel_size * in_channels * out_channels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_bias, (void*)Bias, 1*1 * out_channels * sizeof(float), hipMemcpyHostToDevice);

        float *d_V, *d_U, *d_UV;

        hipMalloc((void**)&d_V, sizeof(float) * in_channels*P*16);
        hipMalloc((void**)&d_U, sizeof(float) * out_channels*in_channels*16);
        hipMalloc((void**)&d_UV, sizeof(float) * out_channels*P*16);
        hipMemcpy(d_U, U, sizeof(float) * out_channels*in_channels*16, hipMemcpyHostToDevice);

        // =================================================执行
        winograd2::calc_V<<<dim3(batch, tile_num, in_channels), dim3(4, 4)>>>(d_A, d_V, P, batch, in_channels, height_A, width_A);
        winograd2::calc_UV<<<dim3(out_channels/2, P/2, 16), dim3(2, 2)>>>(d_U, d_V, d_UV, out_channels, in_channels, P);
        winograd2::calc_AtmA_bias<<<dim3(out_channels, batch, tile_num), dim3(2, 2)>>>(d_UV, d_B, d_bias, out_channels, P, height_B, width_B, tile_num);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_B * height_B * out_channels * sizeof(float), hipMemcpyDeviceToHost);
    }
};


class maxpooling2d {
private:
    int kernel_size;
    int padding;
    int strides;

public:
    maxpooling2d(int kernel_sz, int padding, int strides):
            kernel_size(kernel_sz), padding(padding),strides(strides){}

    void forward(tensor<float>* tensor_A, tensor<float>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width,channels_A=tensor_A->channels;
        float *A=tensor_A->data;
        const int batch = tensor_A->batch;
        //  printf("A: %d %d %d %d \n",height_A,width_A,channels_A,batch);

        // =================================================计算输出大小
        int height_B = (height_A-kernel_size+2*padding)/strides+1;
        int width_B = (width_A-kernel_size+2*padding)/strides+1;

        float* B = (float*)malloc(sizeof(float)*height_B*width_B*channels_A*batch);
        tensor_B=new tensor<float>(B,width_B,height_B,channels_A,batch);

//        printf("B: %d %d %d\n",tensor_B->height,tensor_B->width,tensor_B->channels);
        float* d_A;
        float* d_B;
        hipMalloc((void**)&d_A, batch * width_A * height_A * channels_A * sizeof(float));
        hipMalloc((void**)&d_B, batch * width_B * height_B * channels_A * sizeof(float));

//        printf("cuda malloc ok\n");
        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * channels_A * sizeof(float), hipMemcpyHostToDevice);
//        printf("cuda cpy ok\n");

        // =================================================执行
        int nthreads = batch * width_B * height_B * channels_A;

        int num=nthreads/400+1;
        dim3 blockNum(num, 1);
        dim3 threadsPerBlock(20, 20);

        MaxPoolForward <<<blockNum, threadsPerBlock>>>(d_A,d_B, nthreads, channels_A, height_A, width_A, height_B, width_B,
                                                       kernel_size,kernel_size,strides,strides,padding,padding);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_B * height_B * channels_A *sizeof(float), hipMemcpyDeviceToHost);

        //       printf("Maxpooling done! %d %d %d %d %f\n",tensor_B->batch,tensor_B->channels,tensor_B->height,tensor_B->width,tensor_B->data[0]);

    }

};

class GlobalAvgpooling{
public:
    GlobalAvgpooling()= default;
    void forward(tensor<float>* tensor_A, tensor<float>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width,channels_A=tensor_A->channels;
        float *A=tensor_A->data;
        const int batch = tensor_A->batch;

        // =================================================计算输出大小
        int height_B = 1;
        int width_B = 1;

        float* B = (float*)malloc(sizeof(float)*height_B*width_B*channels_A*batch);
        tensor_B=new tensor<float>(B,width_B,height_B,channels_A,batch);

        float* d_A;
        float* d_B;
        hipMalloc((void**)&d_A, batch * width_A * height_A * channels_A * sizeof(float));
        hipMalloc((void**)&d_B, batch * width_B * height_B * channels_A * sizeof(float));

        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * channels_A * sizeof(float), hipMemcpyHostToDevice);
        // =================================================执行
        int nthreads = batch * width_B * height_B * channels_A;

        int num=nthreads/400+1;
        dim3 blockNum(num, 1);
        dim3 threadsPerBlock(20, 20);

        AvgPoolForward<<<blockNum, threadsPerBlock>>>(d_A,d_B, nthreads,channels_A,height_A,width_A,height_B,width_B,
                                                      height_A, width_A,1,1,0,0);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_B * height_B * channels_A *sizeof(float), hipMemcpyDeviceToHost);

//        printf("Avgpooling done! %d %d %d %d %f\n",tensor_B->batch,tensor_B->channels,tensor_B->height,tensor_B->width,tensor_B->data[0]);

    }
};


class Relu{
public:
    Relu()= default;

    void forward(tensor<float>* tensor_A, tensor<float>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width, channels_A=tensor_A->channels;
        float *A=tensor_A->data;
        const int batch = tensor_A->batch;

        float* B = (float*)malloc(sizeof(float)*height_A*width_A*channels_A*batch);
        tensor_B=new tensor<float>(B,width_A,height_A,channels_A,batch);

        float* d_A;
        float* d_B;
        hipMalloc((void**)&d_A, batch * width_A * height_A * channels_A * sizeof(float));
        hipMalloc((void**)&d_B, batch * width_A * height_A * channels_A * sizeof(float));

        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * channels_A * sizeof(float), hipMemcpyHostToDevice);

        // =================================================执行
        int nthread = width_A * height_A * batch * channels_A;

        int num=nthread/400+1;
        dim3 blockNum(num, 1);
        dim3 threadsPerBlock(20, 20);

        relu <<<blockNum, threadsPerBlock>>>(d_A,d_B,nthread);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_A * height_A * channels_A * sizeof(float), hipMemcpyDeviceToHost);

    }
};


class Add{
public:
    Add()= default;
    // A+B=C
    void forward(tensor<float>* tensor_A, tensor<float>* tensor_B, tensor<float>*& tensor_C) {
        const int height=tensor_A->height, width=tensor_A->width, channels=tensor_A->channels;
        float *A=tensor_A->data;
        float *B=tensor_B->data;
        const int batch = tensor_A->batch;

        float* C = (float*)malloc(sizeof(float)*height*width*channels*batch);
        tensor_C = new tensor<float>(C,width,height,channels,batch);

        float *d_A;
        float *d_B;
        float *d_C;
        hipMalloc((void **) &d_A, batch * width * height * channels * sizeof(float));
        hipMalloc((void **) &d_B, batch * width * height * channels * sizeof(float));
        hipMalloc((void **) &d_C, batch * width * height * channels * sizeof(float));

        hipMemcpy((void *) d_A, (void *) A, batch * width * height * channels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void *) d_B, (void *) B, batch * width * height * channels * sizeof(float), hipMemcpyHostToDevice);

        int nthread = width * height * batch * channels;

        int num=nthread/400+1;
        dim3 blockNum(num, 1);
        dim3 threadsPerBlock(20, 20);

        add<<<blockNum, threadsPerBlock>>>(d_A, d_B, d_C,nthread);

        hipMemcpy((void *) tensor_C->data, (void *) d_C, batch * width * height * channels * sizeof(float), hipMemcpyDeviceToHost);
    }
};


class Gemm{
private:
    int in_dim;
    int out_dim;
    float* Weight; // out_dim x in_dim
    float* Bias; // out_dim
public:
    Gemm(int indim, int outdim, float* weight, float* bias):in_dim(indim),out_dim(outdim),Weight(weight),Bias(bias){}
    // A x Weight + Bias = B
    void forward(tensor<float>* tensor_A, tensor<float>*& tensor_B){
        const int batch = tensor_A->batch;
        float *A=tensor_A->data;

        float* B = (float*)malloc(sizeof(float)*out_dim*batch);
        tensor_B=new tensor<float>(B,1,1,out_dim,batch);

        float* d_A;
        float* d_B;
        float* d_W;
        float* d_bias;
//        printf("start cuda malloc\n");
        hipMalloc((void**)&d_A, batch * in_dim * sizeof(float));
        hipMalloc((void**)&d_B, batch * out_dim * sizeof(float));
        hipMalloc((void**)&d_W, out_dim * in_dim * sizeof(float));
        hipMalloc((void**)&d_bias, out_dim * sizeof(float));

        hipMemcpy((void*)d_A, (void*)A, batch * in_dim * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_W, (void*)Weight, out_dim * in_dim * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_bias, (void*)Bias, out_dim * sizeof(float), hipMemcpyHostToDevice);

        int nthreads = batch * out_dim;

        dim3 blockNum(batch*out_dim/400+1,1);
        dim3 threadsPerBlock(20, 20);

        simple_matmul<<<blockNum, threadsPerBlock>>>(d_A, d_B, d_W, d_bias, nthreads, batch, in_dim, out_dim);

        hipMemcpy((void*)tensor_B->data, (void*)d_B,  batch * out_dim * sizeof(float), hipMemcpyDeviceToHost);

        //      printf("gemm done!: %f %f %d %d %d %d\n",tensor_B->data[0], tensor_B->data[132],
        //             tensor_B->batch, tensor_B->channels,tensor_B->height,tensor_B->width);
    }
};

class BasicBlock{
private:
    float* Weight1;
    float* Bias1;
    float* Weight2;
    float* Bias2;
    conv2d *conv1;
    conv2d *conv2;
    conv_wino_2x2_3x3 *conv1_2x2;
    conv_wino_2x2_3x3 *conv2_2x2;
    conv_wino_4x4_3x3 *conv1_4x4;
    conv_wino_4x4_3x3 *conv2_4x4;
    Relu *relu;
    Add *add;
    int conv_type;

public:
    ~BasicBlock(){};

    BasicBlock(int _inplanes, int _planes, float* weight1, float* bias1, float* weight2, float* bias2, int conv_type):
            Weight1(weight1),Bias1(bias1),Weight2(weight2),Bias2(bias2),conv_type(conv_type)
    {
        if (conv_type == 1){
            conv1 = new conv2d{_inplanes, _planes, Weight1,Bias1, 3, 1, 1, 1};//3*3卷积，stride=1
            conv2 = new conv2d{_planes, _planes, Weight2, Bias2,3, 1, 1, 1};//3*3卷积，stride=1
        }else if (conv_type == 2){
            conv1_2x2 = new conv_wino_2x2_3x3{_inplanes, _planes, Weight1,Bias1, 3, 1, 1, 1};
            conv2_2x2 = new conv_wino_2x2_3x3{_planes, _planes, Weight2, Bias2,3, 1, 1, 1};
        }else if (conv_type == 4){
            conv1_4x4 = new conv_wino_4x4_3x3{_inplanes, _planes, Weight1,Bias1, 3, 1, 1, 1};
            conv2_4x4 = new conv_wino_4x4_3x3{_planes, _planes, Weight2, Bias2,3, 1, 1, 1};
        }

        relu = new Relu{};
        add = new Add{};
    };

    void forward(tensor<float>* A, tensor<float>*& B){
        tensor<float>* residual = new tensor<float>(*A);
        tensor<float> *output, *output2;

        if (conv_type == 1){
            conv1->forward(A,output);
        }else if (conv_type == 2){
            conv1_2x2->forward(A,output);
        }else if (conv_type == 4){
            conv1_4x4->forward(A,output);
        }
//        printf("conv ok %d %d %d %d %f \n", output->batch,output->channels,output->height,output->width, output->data[131]);
        relu->forward(output,output2);
//        printf("relu ok output %d %d %d %d %f \n",output2->batch,output2->channels,output2->height,output2->width, output2->data[131]);
        free(output->data);
        free(output);

        if (conv_type == 1){
            conv2->forward(output2,output);
        }else if (conv_type == 2){
            conv2_2x2->forward(output2,output);
        }else if (conv_type == 4){
            conv2_4x4->forward(output2,output);
        }

//        conv2->forward(output2,output);
//        printf("conv2 ok %d %d %d %d %f \n", output->batch,output->channels,output->height,output->width, output->data[131]);
        free(output2->data);
        free(output2);
//        printf("before add %f %f %f \n",output->data[131],residual->data[131],A->data[131]);
        add->forward(output,residual,output2); //output2=output+residual
//        printf("add ok %d %d %d %d %f \n",output2->batch,output2->channels,output2->height,output2->width, output2->data[131]);
        free(output->data);
        free(output);
        relu->forward(output2,output);
//        printf("relu ok\n");
        free(output2->data);
        free(output2);

        B = output;
        //       printf("Basic block ok: %d %d %d %d\n",B->batch,B->channels,B->height,B->width);
    };
};


class Bottleneck{
private:
    float *Weight1,*Bias1;
    float *Weight2,*Bias2;
    float *Weight3,*Bias3;
    conv2d *conv1,*conv2,*conv3;
    conv_wino_2x2_3x3 *conv1_2x2;
    conv_wino_2x2_3x3 *conv2_2x2;
    conv_wino_4x4_3x3 *conv1_4x4;
    conv_wino_4x4_3x3 *conv2_4x4;
    Relu *relu;
    Add *add;
    int conv_type;

public:
    ~Bottleneck(){};

    Bottleneck(int _inplanes, int _planes, float* weight1, float* bias1, float* weight2, float* bias2, float* weight3, float* bias3,int _stride, int conv_type):
            Weight1(weight1),Bias1(bias1),Weight2(weight2),Bias2(bias2),Weight3(weight3),Bias3(bias3),conv_type(conv_type)
    {
        if (conv_type == 1){
            conv1 = new conv2d{_inplanes,_planes,weight1,bias1,3,1,1,_stride};//3*3卷积 stride=_strinde ic=_inplanes oc=width
            conv2 = new conv2d{_planes,_planes,weight2,bias2, 3, 1, 1, 1};//3*3卷积，stride=1,ic\oc=width,groups=_groups,dilation=_dilation
        }else if (conv_type == 2){
            conv1_2x2 = new conv_wino_2x2_3x3{_inplanes,_planes,weight1,bias1,3, 1, 1, _stride};//3*3卷积 stride=_strinde ic=_inplanes oc=width
            conv2_2x2 = new conv_wino_2x2_3x3{_planes,_planes,weight2,bias2, 3, 1, 1, 1};//3*3卷积，stride=1,ic\oc=width,groups=_groups,dilation=_dilation
        }else if (conv_type == 4){
            conv1_4x4 = new conv_wino_4x4_3x3{_inplanes,_planes,weight1,bias1,3,1,1,_stride};//3*3卷积 stride=_strinde ic=_inplanes oc=width
            conv2_4x4 = new conv_wino_4x4_3x3{_planes,_planes,weight2,bias2, 3, 1, 1, 1};//3*3卷积，stride=1,ic\oc=width,groups=_groups,dilation=_dilation
        }

        conv3 = new conv2d{_inplanes,_planes,weight3,bias3, 1, 1, 0, _stride};//1*1 ic=width,oc=_planes*expansion
        relu = new Relu;
        add = new Add;
    };

    void forward(tensor<float>* A, tensor<float>*& B){
        tensor<float>* identity  = new tensor<float>(*A);
        tensor<float> *output, *output2, *output3;
//        printf("start bottleneck!\n");

        if (conv_type == 1){
            conv1->forward(A,output);
        }else if (conv_type == 2){
            conv1_2x2->forward(A,output);
        }else if (conv_type == 4){
            conv1_4x4->forward(A,output);
        }
//        conv1->forward(A,output);
//        printf("conv ok %d %d %d %d %f \n", output->batch,output->channels,output->height,output->width, output->data[131]);
        relu->forward(output,output2);
//        printf("relu ok output %d %d %d %d %f \n",output2->batch,output2->channels,output2->height,output2->width, output2->data[131]);
        free(output->data);
        free(output);

        if (conv_type == 1){
            conv2->forward(output2,output);
        }else if (conv_type == 2){
            conv2_2x2->forward(output2,output);
        }else if (conv_type == 4){
            conv2_4x4->forward(output2,output);
        }
//        conv2->forward(output2,output);

        conv3->forward(identity,output2);

        add->forward(output2,output,output3); //output3=output+output2
        free(output2->data);
        free(output2);
        free(output->data);
        free(output);
        relu->forward(output3,output);
        free(output3->data);
        free(output3);

        B = output;

        //     printf("Bottle neck ok %d %d %d %d %f \n", B->batch,B->channels,B->height,B->width, B->data[131]);

    };

};