//
// Created by admin on 2021/12/4.
//
#include <iostream>
#include <hip/hip_runtime.h>
#include <string.h>
#include "kernels.cu"

using namespace std;

template<class Dtype> struct tensor{
    Dtype* data;
    int width,height,channels,batch;
    //tensor shape: (batch, channels, width, height)
    tensor(Dtype* d, int w, int h, int c, int batch):data(d),width(w),height(h),channels(c),batch(batch){}
    tensor(const tensor<Dtype> &d){
        data = d.data;
        width = d.width;
        height = d.height;
        batch = d.batch;
    }
    tensor(){}
};

template<class Dtype> class conv2d{
private:
    int in_channels;
    int out_channels;
    int kernel_size;
    int dialations;
    int padding;
    int strides;
    Dtype* Weight;
    Dtype* Bias;

public:
    conv2d(int in_c, int out_c, Dtype* weight, Dtype* bias, const int kernel_sz, const int dialations, const int padding, const int strides):
            in_channels(in_c),out_channels(out_c),Weight(weight),Bias(bias),kernel_size(kernel_sz),dialations(dialations),padding(padding),strides(strides){}
    //input->tensor_A; output->tensor_B
    void forward(const tensor<Dtype>* tensor_A, tensor<Dtype>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width;
        const int batch = tensor_A->batch;
        Dtype *A=tensor_A->data;
//        printf("A: %d %d %d %d %f %f\n",height_A,width_A,in_channels,batch, A[132], A[21]);
//        printf("W: %f %f %f\n",Weight[0],Weight[12],Weight[54]);

        // =================================================计算输出大小
        int height_B = (height_A+2*padding-dialations*(kernel_size-1)-1)/strides + 1;
        int width_B = (width_A+2*padding-dialations*(kernel_size-1)-1)/strides + 1;

        Dtype* B = (float*)malloc(sizeof(float)*height_B*width_B*out_channels*batch);
        tensor_B=new tensor<float>(B,width_B,height_B,out_channels,batch);

//        printf("B: %d %d %d %f\n",tensor_B->height,tensor_B->width,tensor_B->channels,tensor_B->data[0]);

        Dtype* d_A;
        Dtype* d_B;
        Dtype* d_K;
        Dtype* d_bias;
//        printf("start cuda malloc\n");
        hipMalloc((void**)&d_A, batch * width_A * height_A * in_channels * sizeof(float));
        hipMalloc((void**)&d_B, batch * width_B * height_B * out_channels * sizeof(float));
        hipMalloc((void**)&d_K, kernel_size*kernel_size * in_channels * out_channels * sizeof(float));
        hipMalloc((void**)&d_bias, 1*1*out_channels* sizeof(float));

        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * in_channels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_K, (void*)Weight, kernel_size*kernel_size * in_channels * out_channels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_bias, (void*)Bias, 1*1 * out_channels * sizeof(float), hipMemcpyHostToDevice);
//        printf("cuda cpy ok\n");
        // =================================================执行
        int nthreads = batch * width_B * height_B * out_channels;

        dim3 blockNum(batch, out_channels);
        dim3 threadsPerBlock(width_B, height_B);
//        printf("Bias: %f %f \n",Bias[0],Bias[1]);
        ConvolutionForward<<<blockNum, threadsPerBlock>>>(d_A, d_B, d_K, d_bias, nthreads,batch, height_A, width_A, in_channels ,height_B, width_B, out_channels,
                           kernel_size,kernel_size,strides,strides,padding,padding);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_B * height_B * out_channels * sizeof(Dtype), hipMemcpyDeviceToHost);

        printf("conv done! %f %f\n",tensor_B->data[0], tensor_B->data[102]);
    }
};


template<class Dtype> class maxpooling2d {
private:
    int kernel_size;
    int padding;
    int strides;

public:
    maxpooling2d(int kernel_sz, int padding, int strides):
            kernel_size(kernel_sz), padding(padding),strides(strides){}

    void forward(tensor<Dtype>* tensor_A, tensor<Dtype>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width,channels_A=tensor_A->channels;
        Dtype *A=tensor_A->data;
        const int batch = tensor_A->batch;
        printf("A: %d %d %d %d \n",height_A,width_A,channels_A,batch);

        // =================================================计算输出大小
        int height_B = (height_A-kernel_size+2*padding)/strides+1;
        int width_B = (width_A-kernel_size+2*padding)/strides+1;

        Dtype* B = (Dtype*)malloc(sizeof(Dtype)*height_B*width_B*channels_A*batch);
        tensor_B=new tensor<float>(B,width_B,height_B,channels_A,batch);

//        printf("B: %d %d %d\n",tensor_B->height,tensor_B->width,tensor_B->channels);
        Dtype* d_A;
        Dtype* d_B;
        hipMalloc((void**)&d_A, batch * width_A * height_A * channels_A * sizeof(float));
        hipMalloc((void**)&d_B, batch * width_B * height_B * channels_A * sizeof(float));

//        printf("cuda malloc ok\n");
        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * channels_A * sizeof(float), hipMemcpyHostToDevice);
//        printf("cuda cpy ok\n");

        // =================================================执行
        int nthreads = batch * width_B * height_B * channels_A;

        dim3 blockNum(batch, channels_A);
        dim3 threadsPerBlock(width_B, height_B);

        MaxPoolForward<Dtype> <<<blockNum, threadsPerBlock>>>(d_A,d_B, nthreads, channels_A, height_A, width_A, height_B, width_B,
                       kernel_size,kernel_size,strides,strides,padding,padding);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_B * height_B * channels_A *sizeof(float), hipMemcpyDeviceToHost);

        printf("Maxpooling done! %d %d %d %d %f\n",tensor_B->batch,tensor_B->channels,tensor_B->height,tensor_B->width,tensor_B->data[0]);

    }

};

template<class Dtype> class GlobalAvgpooling{
public:
    GlobalAvgpooling()= default;
    void forward(tensor<Dtype>* tensor_A, tensor<Dtype>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width,channels_A=tensor_A->channels;
        Dtype *A=tensor_A->data;
        const int batch = tensor_A->batch;

        // =================================================计算输出大小
        int height_B = 1;
        int width_B = 1;

        Dtype* B = (Dtype*)malloc(sizeof(Dtype)*height_B*width_B*channels_A*batch);
        tensor_B=new tensor<float>(B,width_B,height_B,channels_A,batch);

        Dtype* d_A;
        Dtype* d_B;
        hipMalloc((void**)&d_A, batch * width_A * height_A * channels_A * sizeof(float));
        hipMalloc((void**)&d_B, batch * width_B * height_B * channels_A * sizeof(float));

        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * channels_A * sizeof(float), hipMemcpyHostToDevice);
        // =================================================执行
        int nthreads = batch * width_B * height_B * channels_A;

        dim3 blockNum(batch, channels_A);
        dim3 threadsPerBlock(width_B, height_B);

        AvgPoolForward<<<blockNum, threadsPerBlock>>>(d_A,d_B, nthreads,channels_A,height_A,width_A,height_B,width_B,
                       height_A, width_A,1,1,0,0);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_B * height_B * channels_A *sizeof(float), hipMemcpyDeviceToHost);

//        printf("Avgpooling done! %d %d %d %d %f\n",tensor_B->batch,tensor_B->channels,tensor_B->height,tensor_B->width,tensor_B->data[0]);

    }
};


template<class Dtype> class Relu{
public:
    Relu()= default;

    void forward(tensor<Dtype>* tensor_A, tensor<Dtype>*& tensor_B){
        const int height_A=tensor_A->height, width_A=tensor_A->width, channels_A=tensor_A->channels;
        Dtype *A=tensor_A->data;
        const int batch = tensor_A->batch;

        Dtype* B = (Dtype*)malloc(sizeof(Dtype)*height_A*width_A*channels_A*batch);
        tensor_B=new tensor<float>(B,width_A,height_A,channels_A,batch);

        Dtype* d_A;
        Dtype* d_B;
        hipMalloc((void**)&d_A, batch * width_A * height_A * channels_A * sizeof(Dtype));
        hipMalloc((void**)&d_B, batch * width_A * height_A * channels_A * sizeof(Dtype));

        hipMemcpy((void*)d_A, (void*)A, batch * width_A * height_A * channels_A * sizeof(Dtype), hipMemcpyHostToDevice);

        // =================================================执行
        int nthread = width_A * height_A * batch * channels_A;

        dim3 blockNum(batch, channels_A);
        dim3 threadsPerBlock(width_A, height_A);
        relu<Dtype> <<<blockNum, threadsPerBlock>>>(d_A,d_B,nthread);

        hipMemcpy((void*)tensor_B->data, (void*)d_B, batch * width_A * height_A * channels_A * sizeof(Dtype), hipMemcpyDeviceToHost);

    }
};


template<class Dtype> class Add{
public:
    Add()= default;
    // A+B=C
    void forward(tensor<Dtype>* tensor_A, tensor<Dtype>* tensor_B, tensor<Dtype>*& tensor_C) {
        const int height=tensor_A->height, width=tensor_A->width, channels=tensor_A->channels;
        Dtype *A=tensor_A->data;
        Dtype *B=tensor_B->data;
        const int batch = tensor_A->batch;

        Dtype* C = (Dtype*)malloc(sizeof(Dtype)*height*width*channels*batch);
        tensor_C = new tensor<float>(C,width,height,channels,batch);

        Dtype *d_A;
        Dtype *d_B;
        Dtype *d_C;
        hipMalloc((void **) &d_A, batch * width * height * channels * sizeof(float));
        hipMalloc((void **) &d_B, batch * width * height * channels * sizeof(float));
        hipMalloc((void **) &d_C, batch * width * height * channels * sizeof(float));

        hipMemcpy((void *) d_A, (void *) A, batch * width * height * channels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void *) d_B, (void *) B, batch * width * height * channels * sizeof(float), hipMemcpyHostToDevice);

        int nthread = width * height * batch * channels;
        dim3 blockNum(batch, channels);
        dim3 threadsPerBlock(width, height);

        add<<<blockNum, threadsPerBlock>>>(d_A, d_B, d_C,nthread);

        hipMemcpy((void *) tensor_C->data, (void *) d_C, batch * width * height * channels * sizeof(float), hipMemcpyDeviceToHost);
    }
};


template<class Dtype> class Gemm{
private:
    int in_dim;
    int out_dim;
    Dtype* Weight; // out_dim x in_dim
    Dtype* Bias; // out_dim
public:
    Gemm(int indim, int outdim, Dtype* weight, Dtype* bias):in_dim(indim),out_dim(outdim),Weight(weight),Bias(bias){}
    // A x Weight + Bias = B
    void forward(tensor<Dtype>* tensor_A, tensor<Dtype>*& tensor_B){
        const int batch = tensor_A->batch;
        Dtype *A=tensor_A->data;

        Dtype* B = (float*)malloc(sizeof(float)*out_dim*batch);
        tensor_B=new tensor<float>(B,1,1,out_dim,batch);

        Dtype* d_A;
        Dtype* d_B;
        Dtype* d_W;
        Dtype* d_bias;
//        printf("start cuda malloc\n");
        hipMalloc((void**)&d_A, batch * in_dim * sizeof(float));
        hipMalloc((void**)&d_B, batch * out_dim * sizeof(float));
        hipMalloc((void**)&d_W, out_dim * in_dim * sizeof(float));
        hipMalloc((void**)&d_bias, out_dim * sizeof(float));

        hipMemcpy((void*)d_A, (void*)A, batch * in_dim * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_W, (void*)Weight, out_dim * in_dim * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void*)d_bias, (void*)Bias, out_dim * sizeof(float), hipMemcpyHostToDevice);

        int nthreads = batch * out_dim;

        dim3 blockNum(batch, out_dim);
        dim3 threadsPerBlock(1, 1);

        simple_matmul<<<blockNum, threadsPerBlock>>>(d_A, d_B, d_W, d_bias, nthreads, batch, in_dim, out_dim);

        hipMemcpy((void*)tensor_B->data, (void*)d_B,  batch * out_dim * sizeof(Dtype), hipMemcpyDeviceToHost);

        printf("gemm done!: %f %f %d %d %d %d\n",tensor_B->data[0], tensor_B->data[132],
               tensor_B->batch, tensor_B->channels,tensor_B->height,tensor_B->width);
    }
};

template<class Dtype> class BasicBlock{
private:
    Dtype* Weight1;
    Dtype* Bias1;
    Dtype* Weight2;
    Dtype* Bias2;
    conv2d<Dtype> *conv1,*conv2;
    Relu<Dtype> *relu;
    Add<Dtype> *add;

public:
    ~BasicBlock(){};

    BasicBlock(int _inplanes, int _planes, Dtype* weight1, Dtype* bias1, Dtype* weight2, Dtype* bias2):
            Weight1(weight1),Bias1(bias1),Weight2(weight2),Bias2(bias2)
    {
        conv1 = new conv2d<Dtype>{_inplanes, _planes, Weight1,Bias1, 3, 1, 1, 1};//3*3卷积，stride=1
        relu = new Relu<Dtype>{};
        conv2 = new conv2d<Dtype>{_planes, _planes, Weight2, Bias2,3, 1, 1, 1};//3*3卷积，stride=1
        add = new Add<Dtype>{};
    };

    void forward(tensor<Dtype>* A, tensor<Dtype>*& B){
        tensor<Dtype>* residual = new tensor<Dtype>(*A);
        tensor<Dtype> *output, *output2;

        conv1->forward(A,output);
//        printf("conv ok %d %d %d %d %f \n", output->batch,output->channels,output->height,output->width, output->data[131]);
        relu->forward(output,output2);
//        printf("relu ok output %d %d %d %d %f \n",output2->batch,output2->channels,output2->height,output2->width, output2->data[131]);
        free(output->data);
        free(output);
        conv2->forward(output2,output);
//        printf("conv2 ok %d %d %d %d %f \n", output->batch,output->channels,output->height,output->width, output->data[131]);
        free(output2->data);
        free(output2);
//        printf("before add %f %f %f \n",output->data[131],residual->data[131],A->data[131]);
        add->forward(output,residual,output2); //output2=output+residual
//        printf("add ok %d %d %d %d %f \n",output2->batch,output2->channels,output2->height,output2->width, output2->data[131]);
        free(output->data);
        free(output);
        relu->forward(output2,output);
//        printf("relu ok\n");
        free(output2->data);
        free(output2);

        B = output;
        printf("Basic block ok: %d %d %d %d\n",B->batch,B->channels,B->height,B->width);
    };
};

template<class Dtype> class Bottleneck{
private:
    Dtype *Weight1,*Bias1;
    Dtype *Weight2,*Bias2;
    Dtype *Weight3,*Bias3;
    conv2d<Dtype> *conv1,*conv2,*conv3;
    Relu<Dtype> *relu;
    Add<Dtype> *add;

public:
    ~Bottleneck(){};

    Bottleneck(int _inplanes, int _planes, Dtype* weight1, Dtype* bias1, Dtype* weight2, Dtype* bias2, Dtype* weight3, Dtype* bias3,int _stride):
            Weight1(weight1),Bias1(bias1),Weight2(weight2),Bias2(bias2),Weight3(weight3),Bias3(bias3)
    {
        conv1 = new conv2d<Dtype>{_inplanes,_planes,weight1,bias1,3,1,1,_stride};//3*3卷积 stride=_strinde ic=_inplanes oc=width
        conv2 = new conv2d<Dtype>{_planes,_planes,weight2,bias2,3, 1, 1, 1};//3*3卷积，stride=1,ic\oc=width,groups=_groups,dilation=_dilation
        conv3 = new conv2d<Dtype>{_inplanes,_planes,weight3,bias3, 1, 1, 0, _stride};//1*1 ic=width,oc=_planes*expansion
        relu = new Relu<Dtype>;
        add = new Add<Dtype>;
    };

    void forward(tensor<Dtype>* A, tensor<Dtype>*& B){
        tensor<Dtype>* identity  = new tensor<Dtype>(*A);
        tensor<Dtype> *output, *output2, *output3;
//        printf("start bottleneck!\n");
        conv1->forward(A,output);
//        printf("conv ok %d %d %d %d %f \n", output->batch,output->channels,output->height,output->width, output->data[131]);
        relu->forward(output,output2);
//        printf("relu ok output %d %d %d %d %f \n",output2->batch,output2->channels,output2->height,output2->width, output2->data[131]);
        free(output->data);
        free(output);
        conv2->forward(output2,output);

        conv3->forward(identity,output2);

        add->forward(output2,output,output3); //output3=output+output2
        free(output2->data);
        free(output2);
        free(output->data);
        free(output);
        relu->forward(output3,output);
        free(output3->data);
        free(output3);

        B = output;

        printf("Bottle neck ok %d %d %d %d %f \n", B->batch,B->channels,B->height,B->width, B->data[131]);

    };

};
