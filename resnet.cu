#include "hip/hip_runtime.h"
#include ""
#include "block.cu"
#include "string.h"
#include <iostream>
#include "stdio.h"
#include <map>

using namespace std;

void print_tensor(float* Ts, int batch, int channels, int height, int width){
    for(int i=0;i<batch;i++){
        for(int j=0;j<channels;j++){
            for(int k=0;k<height;k++){
                for(int t=0;t<width;t++){
                    printf("%f ",Ts[i*(channels*width*height)+j*(width*height)+k*width+t]);
                }
                printf("\n");
//                break;
            }
            printf("\n");
//            break;
        }
        printf("\n");
    }
}

class Resnet18{
private:
    conv_im2col *conv1;
    Relu *relu;
    maxpooling2d *maxpool;
    GlobalAvgpooling *avgpool;
    Gemm *gemm;
    BasicBlock *layer1,*layer2,*layer3,*layer4,*layer5;
    Bottleneck *neck_layer1,*neck_layer2,*neck_layer3;
    map<string, float*> Parameters;
    hipStream_t stream1;

public:
    Resnet18(map<string, float*> param):Parameters(param){
//        hipStreamCreate(&stream1);
        hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);
        conv1 = new conv_im2col{3,64,Parameters["193"],Parameters["194"], NULL, 7,1,3,2};
        relu = new Relu{};
        maxpool = new maxpooling2d{3,1,2};
        layer1 = new BasicBlock{64,64,Parameters["196"],Parameters["197"],Parameters["199"],Parameters["200"],4};
        layer2 = new BasicBlock{64,64,Parameters["202"],Parameters["203"],Parameters["205"],Parameters["206"],4};
        neck_layer1 = new Bottleneck{64,128,Parameters["208"],Parameters["209"],Parameters["211"],Parameters["212"],Parameters["214"],Parameters["215"],&stream1,2,4};
        layer3 = new BasicBlock{128,128,Parameters["217"],Parameters["218"],Parameters["220"],Parameters["221"],4};
        neck_layer2 = new Bottleneck{128,256,Parameters["223"],Parameters["224"],Parameters["226"],Parameters["227"],Parameters["229"],Parameters["230"],&stream1,2,4};
        layer4 = new BasicBlock{256,256,Parameters["232"],Parameters["233"],Parameters["235"],Parameters["236"],4};
        neck_layer3 =new Bottleneck{256,512,Parameters["238"],Parameters["239"],Parameters["241"],Parameters["242"],Parameters["244"],Parameters["245"],&stream1,2,2};
        layer5 = new BasicBlock{512,512,Parameters["247"],Parameters["248"],Parameters["250"],Parameters["251"],2};
        // all the conv use im2col
//        layer1 = new BasicBlock{64,64,Parameters["196"],Parameters["197"],Parameters["199"],Parameters["200"],1};
//        layer2 = new BasicBlock{64,64,Parameters["202"],Parameters["203"],Parameters["205"],Parameters["206"],1};
//        neck_layer1 = new Bottleneck{64,128,Parameters["208"],Parameters["209"],Parameters["211"],Parameters["212"],Parameters["214"],Parameters["215"],&stream1,2,1};
//        layer3 = new BasicBlock{128,128,Parameters["217"],Parameters["218"],Parameters["220"],Parameters["221"],1};
//        neck_layer2 = new Bottleneck{128,256,Parameters["223"],Parameters["224"],Parameters["226"],Parameters["227"],Parameters["229"],Parameters["230"],&stream1,2,1};
//        layer4 = new BasicBlock{256,256,Parameters["232"],Parameters["233"],Parameters["235"],Parameters["236"],1};
//        neck_layer3 =new Bottleneck{256,512,Parameters["238"],Parameters["239"],Parameters["241"],Parameters["242"],Parameters["244"],Parameters["245"],&stream1,2,1};
//        layer5 = new BasicBlock{512,512,Parameters["247"],Parameters["248"],Parameters["250"],Parameters["251"],1};
//        avgpool = new GlobalAvgpooling{};
//        gemm = new Gemm{512,1000,Parameters["fc.weight"],Parameters["fc.bias"]};

    }

    void forward(float* tensor_A, int height_A, int width_A, int channel_A, int batch,
                 float*& tensor_B, int& height_B, int &width_B, int &channel_B){
        float *tmp_out1,*tmp_out2;
        int height1, width1, channel1;
        int height2, width2, channel2;

//        float Onetime;
//        hipEvent_t start, stop;
//        hipEventCreate(&start);
//        hipEventCreate(&stop);
//        hipEventRecord(start, 0);

        float* A;
        float* B;
        hipMalloc((void**)&A, batch * width_A * height_A * channel_A * sizeof(float));
        hipMemcpy((void*)A, (void*)tensor_A, batch * width_A * height_A * channel_A * sizeof(float), hipMemcpyHostToDevice);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("Memcpy time: %f\n", Onetime);

//        hipEventRecord(start, 0);

        conv1->forward(A, height_A, width_A, channel_A, batch,
                       tmp_out1, height1, width1, channel1);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("conv1 time: %f\n", Onetime);

//        hipEventRecord(start, 0);

        relu->forward(tmp_out1, height1, width1, channel1, batch,
                      tmp_out2, height2, width2, channel2);

        hipFree(tmp_out1);
        maxpool->forward(tmp_out2, height2, width2, channel2, batch,
                         tmp_out1, height1, width1, channel1);
        hipFree(tmp_out2);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("relu+maxpooling time: %f\n", Onetime);

//        hipEventRecord(start, 0);

        layer1->forward(tmp_out1, height1, width1, channel1, batch,
                        tmp_out2, height2, width2, channel2);
        hipFree(tmp_out1);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("basic block1 time: %f\n", Onetime);

//        hipEventRecord(start, 0);

        layer2->forward(tmp_out2, height2, width2, channel2, batch,
                        tmp_out1, height1, width1, channel1);
        hipFree(tmp_out2);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("basic block2 time: %f\n", Onetime);

//        hipEventRecord(start, 0);

        neck_layer1->forward(tmp_out1, height1, width1, channel1, batch,
                             tmp_out2, height2, width2, channel2);
        hipFree(tmp_out1);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("bottleneck1 time: %f\n", Onetime);
//
//        hipEventRecord(start, 0);

        layer3->forward(tmp_out2, height2, width2, channel2, batch,
                        tmp_out1, height1, width1, channel1);
//        printf("layer3\n");
        hipFree(tmp_out2);
        neck_layer2->forward(tmp_out1, height1, width1, channel1, batch,
                             tmp_out2, height2, width2, channel2);
        hipFree(tmp_out1);
        layer4->forward(tmp_out2, height2, width2, channel2, batch,
                        tmp_out1, height1, width1, channel1);
//        printf("layer4\n");
        hipFree(tmp_out2);
        neck_layer3->forward(tmp_out1, height1, width1, channel1, batch,
                             tmp_out2, height2, width2, channel2);
        hipFree(tmp_out1);
        layer5->forward(tmp_out2, height2, width2, channel2, batch,
                        tmp_out1, height1, width1, channel1);

        hipFree(tmp_out2);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("layer3 4 5 time: %f\n", Onetime);

//        hipEventRecord(start, 0);

        avgpool->forward(tmp_out1, height1, width1, channel1, batch,
                         tmp_out2, height2, width2, channel2);
        hipFree(tmp_out1);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("avgpooling time: %f\n", Onetime);

//        hipEventRecord(start, 0);

        gemm->forward(tmp_out2, height2, width2, channel2, batch,
                      B, height_B, width_B, channel_B);

//        hipDeviceSynchronize();
//        hipEventRecord(stop, 0);
//        hipEventSynchronize(stop);
//        hipEventElapsedTime(&Onetime, start, stop);
//        printf("gemm time: %f\n", Onetime);

        tensor_B = (float*)malloc( sizeof(float)*height_B*width_B*channel_B*batch );
        hipMemcpy((void*)tensor_B, (void*)B, batch * width_B * height_B * channel_B * sizeof(float), hipMemcpyDeviceToHost);

    }

};


