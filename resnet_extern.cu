#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "block.cu"
#include "string.h"
#include "stdio.h"
#include <map>

namespace resnet{
    conv_im2col*conv1;
    maxpooling2d *maxpool;
    GlobalAvgpooling *avgpool;
    Gemm *gemm;
    Relu* relu;
    BasicBlock *layer1,*layer2,*layer3,*layer4,*layer5;
    Bottleneck *neck_layer1,*neck_layer2,*neck_layer3;

    void resnet_init(map<string, float*> Parameters){
        conv1 = new conv_im2col{3,64,Parameters["193"],Parameters["194"], true,true, 7,1,3,2};
        relu = new Relu{};
        maxpool = new maxpooling2d{3,1,2};
        layer1 = new BasicBlock{64,64,Parameters["196"],Parameters["197"],Parameters["199"],Parameters["200"],1};
        layer2 = new BasicBlock{64,64,Parameters["202"],Parameters["203"],Parameters["205"],Parameters["206"],1};
        neck_layer1 = new Bottleneck{64,128,Parameters["208"],Parameters["209"],Parameters["211"],Parameters["212"],Parameters["214"],Parameters["215"],2,1};
        layer3 = new BasicBlock{128,128,Parameters["217"],Parameters["218"],Parameters["220"],Parameters["221"],1};
        neck_layer2 = new Bottleneck{128,256,Parameters["223"],Parameters["224"],Parameters["226"],Parameters["227"],Parameters["229"],Parameters["230"],2,1};
        layer4 = new BasicBlock{256,256,Parameters["232"],Parameters["233"],Parameters["235"],Parameters["236"],1};
        neck_layer3 =new Bottleneck{256,512,Parameters["238"],Parameters["239"],Parameters["241"],Parameters["242"],Parameters["244"],Parameters["245"],2,1};
        layer5 = new BasicBlock{512,512,Parameters["247"],Parameters["248"],Parameters["250"],Parameters["251"],1};
        avgpool = new GlobalAvgpooling{};
        gemm = new Gemm{512,1000,Parameters["fc.weight"],Parameters["fc.bias"]};
    }


    void resnet_forward(float* tensor_A, int height_A, int width_A, int channel_A, int batch,
                        float*& tensor_B, int& height_B, int &width_B, int &channel_B){
        float *tmp_out1,*tmp_out2;
        int height1, width1, channel1;
        int height2, width2, channel2;

        hipError_t cudaStatus;

        float* A;
        float* B;
        cudaStatus = hipMalloc((void**)&A, batch * width_A * height_A * channel_A * sizeof(float));
        if (cudaStatus != hipSuccess) {
            printf("malloc A failed\n");
        }
        cudaStatus = hipMemcpy((void*)A, (void*)tensor_A, batch * width_A * height_A * channel_A * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            printf("memcpy A failed\n");
        }
//        printf("======= forward begin =======!\n");
        conv1->forward(A, height_A, width_A, channel_A, batch,
                       tmp_out2, height2, width2, channel2);

        hipFree(A);

//        printf("before maxpooling %d %d %d %d\n",batch, height2, width2, channel2);
        maxpool->forward(tmp_out2, height2, width2, channel2, batch,
                         tmp_out1, height1, width1, channel1);

//        printf("after maxpooling %d %d %d %d \n",
//               batch, channel1,height1,width1);

        hipFree(tmp_out2);
//        printf("======== stage 1==========\n");
        layer1->forward(tmp_out1, height1, width1, channel1, batch,
                        tmp_out2, height2, width2, channel2);
//        printf("layer1\n");
        hipFree(tmp_out1);
        layer2->forward(tmp_out2, height2, width2, channel2, batch,
                        tmp_out1, height1, width1, channel1);
//        printf("layer2\n");
        hipFree(tmp_out2);
        neck_layer1->forward(tmp_out1, height1, width1, channel1, batch,
                             tmp_out2, height2, width2, channel2);
        hipFree(tmp_out1);
        layer3->forward(tmp_out2, height2, width2, channel2, batch,
                        tmp_out1, height1, width1, channel1);
//        printf("layer3\n");
        hipFree(tmp_out2);
        neck_layer2->forward(tmp_out1, height1, width1, channel1, batch,
                             tmp_out2, height2, width2, channel2);
        hipFree(tmp_out1);
        layer4->forward(tmp_out2, height2, width2, channel2, batch,
                        tmp_out1, height1, width1, channel1);

        hipFree(tmp_out2);
        neck_layer3->forward(tmp_out1, height1, width1, channel1, batch,
                             tmp_out2, height2, width2, channel2);
        hipFree(tmp_out1);
        layer5->forward(tmp_out2, height2, width2, channel2, batch,
                        tmp_out1, height1, width1, channel1);

        hipFree(tmp_out2);
//        printf("========= stage 2 ===========\n");
//        printf("before avg %f %f %d %d %d %d\n",tmp_out1->data[0], tmp_out1->data[2],
//               tmp_out1->batch, tmp_out1->channels,tmp_out1->height,tmp_out1->width);
        avgpool->forward(tmp_out1, height1, width1, channel1, batch,
                         tmp_out2, height2, width2, channel2);
//        printf("after avg: %f %f %d %d %d %d\n",tmp_out2->data[0], tmp_out2->data[2],
//               tmp_out2->batch, tmp_out2->channels,tmp_out2->height,tmp_out2->width);
//        print_tensor<float>(tmp_out2);
        hipFree(tmp_out1);
        gemm->forward(tmp_out2, height2, width2, channel2, batch,
                      B, height_B, width_B, channel_B);
        hipFree(tmp_out2);

//        printf("after gemm: %d %d %d %d \n", height_B, width_B, channel_B, batch);
//        tensor_B = (float*)malloc( sizeof(float)*height_B*width_B*channel_B*batch );
        hipMemcpy((void*)tensor_B, (void*)B, batch * width_B * height_B * channel_B * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(B);
    }
}

extern "C" void init(map<string, float*> Parameters){
       resnet::resnet_init(Parameters);
}

extern "C" void forward(float* tensor_A, int height_A, int width_A, int channel_A, int batch,
                        float*& tensor_B, int& height_B, int &width_B, int &channel_B){
    resnet::resnet_forward(tensor_A, height_A, width_A, channel_A, batch,
                        tensor_B, height_B, width_B, channel_B);
}
