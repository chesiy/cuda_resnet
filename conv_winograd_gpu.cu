#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <math.h>

void serial_matmul(float* A0, float*B0, float*C0, 
    int dim_1, int dim_2, int dim_3){
    // A: dim1 x dim2, B: dim2 x dim3, C: dim1 x dim3
    for(int i=0; i<dim_1; i++){
        for(int j=0; j<dim_3; j++){
            float tmp = 0;
            for(int k=0; k<dim_2; k++){
                tmp += A0[i*dim_2+k] * B0[k*dim_3+j];
            }
            C0[i*dim_3+j] = tmp;
        }
    }
}

float G[12] = {
    1, 0, 0,
    0.5, 0.5, 0.5,
    0.5, -0.5, 0.5,
    0, 0, 1
};

float G_T[12] = {
    1, 0.5, 0.5, 0,
    0, 0.5, -0.5, 0,
    0, 0.5, 0.5, 1
};

float B_T[16] = {
    1, 0, -1, 0,
    0, 1, 1, 0,
    0, -1, 1, 0,
    0, 1, 0, -1
};
    
float B[16] = {
    1, 0, 0, 0,
    0, 1, -1, 1, 
    -1, 1, 1, 0,
    0, 0, 0, -1
};

float A_T[8] = {
    1, 1, 1, 0,
    0, 1, -1, -1
};

float A[8] = {
    1, 0,
    1, 1, 
    1, -1,
    0, -1
};


void calc_GgGt(float*g, float*out){
    // G: 4x3, g: 3x3
    float tmp1[12];
    serial_matmul(G, g, tmp1, 4, 3, 3);
    serial_matmul(tmp1, G_T, out, 4, 3, 4);
}

void calc_BtdB(float*d, float*out){
    // B: 4x4
    float tmp1[16];
    serial_matmul(B_T, d, tmp1, 4, 4, 4);
    serial_matmul(tmp1, B, out, 4, 4, 4);
    return;
}

void calc_AtmA(float*m, float*out){
    float tmp1[8];
    serial_matmul(A_T, m, tmp1, 2, 4, 4);
    serial_matmul(tmp1, A, out, 2, 4, 2);
    return;
}


void matmul_4x4(float* U, float* V, float* M, int out_channels, int in_channels, int P){
    // U: out_channnels x in channels x 16, V: in_channels x P x 16, M: out_channels x P x 16
    for(int t=0; t<16; t++){
        for(int i=0; i<out_channels; i++){
            for(int j=0; j<P; j++){
                float tmp = 0;
                for(int k=0; k<in_channels; k++){
                    tmp += U[i*in_channels*16 + k*16 + t] * V[k*P*16 + j*16 + t];
                }
                M[i*P*16 + j*16 + t] = tmp;
            }
        }
    }
}

void calc_U(float* kernel, float*U, int in_channels, int out_channels){
    for(int k=0; k<out_channels; k++){
        for(int c=0; c<in_channels; c++){
            float* g_kc = kernel + (k*in_channels*9 + c*9); // kernel[k, c]
            float u[16];
            calc_GgGt(g_kc, u);
            for(int i=0; i<16; i++){
                U[k*in_channels*16 + c*16 + i] = u[i];
            }
        }
    }
}

__global__ void calc_V(float* inp, float* V, int P, int batch_size, int in_channels, int in_numrow, int in_numcol){
    // each block has 16 threads, and in total P*in_channels=(batch_size*tile_num*in_channels) blocks
    __shared__ float inp_shared[4][4];
    __shared__ float Btd_shared[4][4];

    // TODO: OPTIMIZE THIS
    int cur_batch = blockIdx.x, cur_channel = blockIdx.z;
    int cur_row = blockIdx.y / in_numcol -1 + threadIdx.x; // tile_num * 2 - 1 + threadIdx.x
    int cur_col = blockIdx.y % in_numcol -1 + threadIdx.y;

    if(cur_row >= 0 && cur_row < in_numrow && cur_col >= 0 && cur_col < in_numcol) 
        inp_shared[threadIdx.x][threadIdx.y] = 
            inp[cur_batch*in_channels*in_numrow*in_numcol + cur_batch*in_numrow*in_numcol + cur_row*in_numcol + cur_col];
    else
        inp_shared[threadIdx.x][threadIdx.y] = 0;

    __syncthreads();

    // Btd
    switch(threadIdx.x){
        case 0:
            Btd_shared[threadIdx.x][threadIdx.y] = inp_shared[0][threadIdx.y] - inp_shared[2][threadIdx.y];
        case 1:
            Btd_shared[threadIdx.x][threadIdx.y] = inp_shared[1][threadIdx.y] + inp_shared[2][threadIdx.y];
        case 2:
            Btd_shared[threadIdx.x][threadIdx.y] = -inp_shared[1][threadIdx.y] + inp_shared[2][threadIdx.y];
        case 3:
            Btd_shared[threadIdx.x][threadIdx.y] = inp_shared[1][threadIdx.y] - inp_shared[3][threadIdx.y];
    }
    __syncthreads();

    // BtdB
    float tmp = 0;
    switch(threadIdx.y){
        case 0:
            tmp = Btd_shared[threadIdx.x][0] - Btd_shared[threadIdx.x][2];
        case 1:
            tmp = Btd_shared[threadIdx.x][1] + Btd_shared[threadIdx.x][2];
        case 2:
            tmp = -Btd_shared[threadIdx.x][1] + Btd_shared[threadIdx.x][2];
        case 3:
            tmp = Btd_shared[threadIdx.x][1] - Btd_shared[threadIdx.x][3];
    }
    __syncthreads();

    // V[cur_channel, b, threadIdx.x, threadIdx.y] = tmp, and b = (blockIdx.x*in_numrow*in_numcol)/4 + blockIdx.y
    V[cur_channel*P*16 + blockIdx.x*in_numrow*in_numcol*4+blockIdx.y*16 + threadIdx.x*4 + threadIdx.y] = tmp;

    __syncthreads(); // ? before call next function, many synch simultaneously ?
}


__global__ void calc_UV(float* U, float* V, float* out, int out_channels, int in_channels, int P, int mm_tilewidth){
    // U: out_channels x in_channels x 16, V: in_channels x P x 16, out: out_channels x P x 16
    __shared__ float Uds[mm_tilewidth][mm_tilewidth];
    __shared__ float Vds[mm_tilewidth][mm_tilewidth];
    
    int row = blockIdx.x * mm_tilewidth + threadIdx.x;
    int col = blockIdx.y * mm_tilewidth + threadIdx.y;
    int place_in_16 = blockIdx.z;
    float p_value = 0;

    for(int m=0; m<in_channels/mm_tilewidth; m++){
        // U[row, m*mm_tilewidth+threadIdx.y, place_in_16]
        Uds[threadIdx.x][threadIdx.y] = U[row*mm_tilewidth*16 + (m*mm_tilewidth+threadIdx.y)*16 + place_in_16]; 
        // V[m*mm_tilewidth+threadIdx.x, col, place_in_16]
        Vds[threadIdx.x][threadIdx.y] = V[(m*mm_tilewidth+threadIdx.x)*mm_tilewidth*16 + col*16 + place_in_16];
        __syncthreads();

        for(int k=0; k<mm_tilewidth; k++){
            p_value += Uds[threadIdx.x][k] + Vds[k][threadIdx.y];
        }
        __syncthreads();
    }

    out[row*P*16 + col*16 + place_in_16] = p_value;
}

__global__ void calc_AtmA(float* M, float* out, int out_channels, int P, int out_numrow, int out_numcol, int tile_num){
    // each block has 4 threads, and in total out_channels*P=(out_channels*batch_size*tile_num) blocks
    int cur_channel = blockIdx.x, cur_batch = blockIdx.y;
    int cur_tilerow = blockIdx.z / (out_numcol/2), cur_tilecol = blockIdx.z % (out_numcol/2);

    __shared__ float m[4][4];
    __shared__ float Atm[2][4];
    m[threadIdx.x][threadIdx.y] = M[cur_channel*P*16 + (cur_batch*tile_num+blockIdx.z)*16 + threadIdx.x*4 + threadIdx.y];
    m[threadIdx.x][threadIdx.y+2] = M[cur_channel*P*16 + (cur_batch*tile_num+blockIdx.z)*16 + threadIdx.x*4 + threadIdx.y + 2];
    m[threadIdx.x+2][threadIdx.y] = M[cur_channel*P*16 + (cur_batch*tile_num+blockIdx.z)*16 + (threadIdx.x+2)*4 + threadIdx.y];
    m[threadIdx.x+2][threadIdx.y+2] = M[cur_channel*P*16 + (cur_batch*tile_num+blockIdx.z)*16 + (threadIdx.x+2)*4 + threadIdx.y + 2];
    __syncthreads();

    switch(threadIdx.x){
        case 0:
            Atm[threadIdx.x][threadIdx.y] = m[threadIdx.y][0] + m[threadIdx.y][1] + m[threadIdx.y][2];
            Atm[threadIdx.x][threadIdx.y+2] = m[threadIdx.y+2][0] + m[threadIdx.y+2][1] + m[threadIdx.y+2][2];
        case 1:
            Atm[threadIdx.x][threadIdx.y] = m[threadIdx.y][0] - m[threadIdx.y][1] - m[threadIdx.y][2];
            Atm[threadIdx.x][threadIdx.y+2] = m[threadIdx.y+2][0] - m[threadIdx.y+2][1] - m[threadIdx.y+2][2];
    }
    __syncthreads();

    float tmp = 0;
    switch(threadIdx.y){
        case 0:
            tmp = Atm[threadIdx.x][0] + Atm[threadIdx.x][1] + Atm[threadIdx.x][2];
        case 1:
            tmp = Atm[threadIdx.x][1] - Atm[threadIdx.x][2] - Atm[threadIdx.x][3];
    }
    __syncthreads();

    // out[cur_batch, cur_channel, cur_tilerow*2+threadIdx.x, cur_tilecol*2+threadIdx.y]
    out[cur_batch*out_channels*out_numrow*out_numcol + cur_channel*out_numrow*out_numcol + 
            (2*cur_tilerow+threadIdx.x)*out_numcol + 2*cur_tilecol+threadIdx.y] = tmp; 

}


int main()
{  
    // EASY CASE
    int in_channels=2, out_channels=1, inp_row=2, inp_col=2, P=1;
    float kernel[18], input[8], output[4]; // kernel: 1*2*3*3, input: 1*2*2*2, output: 1*1*2*2
    for(int i=0; i<18; i++) kernel[i] = i+1;
    for(int i=0; i<8; i++) input[i] = i+1;

    float *d_kernel, *d_inp, *d_out;

    hipMalloc((void**)&d_kernel, sizeof(float) * 18);
    hipMalloc((void**)&d_inp, sizeof(float) * 8);
    hipMalloc((void**)&d_out, sizeof(float) * 4);

    hipMemcpy(d_kernel, kernel, sizeof(float) * 18, hipMemcpyHostToDevice);
    hipMemcpy(d_inp, input, sizeof(float) * 8, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(10, 20, 1);
    dim3 threadsPerBlock(1, 1, 1);

    float U[32]; // out_channel(1)*in_channel(2)*16
    float *V;
    hipMalloc((void**)&V, sizeof(float) * out_channels*P*16);

    calc_U(kernel, U, in_channels, out_channels); // CPU function, as it can be calculated beforehand
    
    for(int i=0; i<1; i++){
        for(int j=0; j<1; j++){
            for(int k=0; k<2; k++){
                for(int l=0; l<2; l++){
                    float now_element = output[i*4 + j*4 + k*2 + l];
                    printf("%f ", now_element);
                }
                printf(" \n");
            }
            printf(" \n");
        }
    }


    // HARD CASE
    // float kernel[72], input[144], output[288]; // kernel: 4*2*3*3, input: 2*2*6*6, output: 2*4*6*6
    // for(int i=0; i<72; i++) kernel[i] = i;
    // for(int i=0; i<144; i++) input[i] = i;
    
    // printf("start testing\n");
    // batch_trivial_conv2d_2x2_3x3(input, kernel, output,
    //     2, 6, 6, 2, 4, 1);
    
    // for(int i=0; i<2; i++){
    //     for(int j=0; j<4; j++){
    //         for(int k=0; k<6; k++){
    //             for(int l=0; l<6; l++){
    //                 float now_element = output[i*144 + j*36 + k*6 + l];
    //                 printf("%f ", now_element);
    //             }
    //             printf(" \n");
    //         }
    //         printf(" \n");
    //     }
    // }
    return 0;
}